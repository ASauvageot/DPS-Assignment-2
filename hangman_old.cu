#include <iostream>
#include <cstdlib>
#include <ctime>
#include <string>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
 using namespace chrono;


const int MAX_TRIES = 5;
#define WORD_SIZE 1048576

void init_zero(int* a, int n) {
 for (int i = 0; i < n; i++)
	a[i] = 0;
}

void init_null(char* a, int n) {
 for (int i = 0; i < n; i++)
	a[i] = '\0';
}

int letterFill(char, string, string&);

 void reportTime(const char* msg, steady_clock::duration span) { 

    double nsecs = double(span.count()) *
    steady_clock::period::num / steady_clock::period::den;
    std::cout << std::fixed;
    std::cout << msg << " - took - " <<
    nsecs << " secs" << std::endl;

 }
 
 
//gets matches, and edits strings..
__global__ void searchLetter(char* empty, char* word, char* guess, int* count, int* fcount, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//if guessed letter is the letter at word[i]
	if (guess[0] == word[i]){
		//Add to count
		count[i] = 1;
		//Edit empty to have letter filled in.
		empty[i] = word[i];
	}
	else{
		count[i] = 0;
	}
	__syncthreads();
	
	for (int stride = 1; stride < n; stride *= 2) {
		if (i % (2 * stride) == 0)
			count[i] += count[i + stride];
		__syncthreads();
	}
	if (threadIdx.x == 0)
         fcount[blockIdx.x] = count[i];
}

__global__ void count_final(int* fcount, int n) {

	int i = threadIdx.x;

	for (int stride = 1; i + stride < n; stride *= 2) {
         if (i % (2 * stride) == 0)
             fcount[i] += fcount[i + stride];
         __syncthreads();
    }
	
}

int main()
{
	string name;
	char letter;
	int num_of_wrong_guesses = 0;
	string word;

	//choose and copy a word from array of words randomly
	srand(time(NULL));
	int n = rand() % 10;

	//test data to make load slow.
	word = "";
	int l;
	int fail1 = rand() % 26;
	int fail2 = rand() % 26;
	int fail3 = rand() % 26;
	int fail4 = rand() % 26;
	int super = rand() % 26;

	for (l = 0; l<WORD_SIZE; l++){

		int ran = rand() % 26;

		if (ran == fail1 || ran == fail2 || ran == fail3 || ran == fail4){

		}
		else if (ran == super){
			char cch = 'a' + ran;
			word += cch;
			word += cch;
			word += cch;
		}
		else{
			char cch = 'a' + ran;
			word += cch;
		}
	}

	int d;
	hipDeviceProp_t prop;
	hipGetDevice(&d);
	hipGetDeviceProperties(&prop, d);
	int ntpb_x = prop.maxThreadsDim[0];
	cout << prop.major << prop.minor << endl;
	int nblks =  (WORD_SIZE + ntpb_x - 1) / ntpb_x; // number of blocks

		
	// Initialize the secret word with the * character.
	string unknown(word.length(), '*');
	// welcome the user
	cout << "\n\nWelcome to Letter Search...Guess a Letter!";
	cout << "\n\nEach letter is represented by a star.";
	cout << "\n\nYou have to type only one letter in one try";
	cout << "\n\nYou have " << MAX_TRIES << " tries to try and guess the word.";
	cout << "\n~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~";
	// Loop until the guesses are used up
	while (num_of_wrong_guesses < MAX_TRIES)
	{
		
		
		//Uncomment below for better game play.
		//cout << "\n\n" << unknown;
		cout << "\n\nGuess a letter: ";
		cin >> letter;
		// Fill secret word with letter if the guess is correct,
		// otherwise increment the number of wrong guesses.

		//TODO
		    steady_clock::time_point ts, te;
		std::srand(std::time(nullptr));
		ts = steady_clock::now();


		int n = word.length();

		//MEMaloc d_empty, d_word, d_guess, d_count

		char* d_empty;
		hipMalloc((void**)&d_empty, nblks * ntpb_x*sizeof(char));

		char*d_word;
		hipMalloc((void**)&d_word, nblks * ntpb_x*sizeof(char));

		char* d_guess;
		hipMalloc((void**)&d_guess, sizeof(char));

		int* d_count;
		hipMalloc((void**)&d_count, nblks * ntpb_x*sizeof(int));

		int* d_fcount;
		hipMalloc((void**)&d_fcount, nblks*sizeof(int));
		
		char* wordchar = new char[nblks * ntpb_x];
		init_null( wordchar, nblks * ntpb_x);
		char* emptychar = new char[nblks * ntpb_x];
		init_null( emptychar, nblks * ntpb_x);
		char* guesschar = new char[sizeof(char)];
		int * h_count = new int[nblks * ntpb_x];
		init_zero(h_count, nblks * ntpb_x);
		int * h_fcount = new int[nblks];


		//int lets = 0; not needed
		//this should go to kernal.
		//lets = letterFill(letter, word, unknown);
		//End going to kernel.

		//TODO
		

		//MEMCPY all above to device

		//first put stings into char array

		memcpy(wordchar, word.c_str(), word.length() + 1);
		memcpy(emptychar, unknown.c_str(), unknown.length() + 1);
		guesschar = &letter;

		//Copy char arrays into cuda.
		hipMemcpy(d_empty, emptychar, nblks * ntpb_x * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_word, wordchar, nblks * ntpb_x * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_guess, guesschar, sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_count, h_count, nblks * ntpb_x * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_fcount, h_count, nblks * sizeof(int), hipMemcpyHostToDevice);



		searchLetter <<<nblks, ntpb_x>>>(d_empty, d_word, d_guess, d_count, d_fcount, ntpb_x);
		count_final <<<1, nblks>>>(d_fcount, nblks);

		//reverse above steps.

		hipMemcpy(emptychar, d_empty, n * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(wordchar, d_word, n * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(guesschar, d_guess, sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(h_count, d_fcount, nblks*sizeof(int), hipMemcpyDeviceToHost);

		//copied back to chars, now copy to strings
		unknown = emptychar;

		te = steady_clock::now();
		reportTime("Search Time: ", te - ts); 
		
		int final_count = h_count[0];
		if (final_count == 0)
		{
			cout << endl << "Whoops! That letter isn't in there!" << endl;
			num_of_wrong_guesses++;
		}
		else
		{
			cout << endl << "You found " << final_count << " letters! Isn't that exciting!" << endl;
		}
		// Tell user how many guesses has left.
		cout << "You have " << MAX_TRIES - num_of_wrong_guesses;
		cout << " guesses left." << endl;
		// Check if user guessed the word.
		if (word == unknown)
		{
			cout << word << endl;
			cout << "Yeah! You got it!";
			break;
		}
	
	// TODO
	// cuda free var's
	 hipFree(d_empty);
	 hipFree(d_word);
	 hipFree(d_guess);
	 hipFree(d_count);
	}



	if (num_of_wrong_guesses == MAX_TRIES)
	{
		cout << "\nSorry, you lose...you've been hanged." << endl;
		cout << "The word was : " << word << endl;
	}
	cin.ignore();
	cin.get();
	return 0;
}
/* Take a one character guess and the secret word, and fill in the
unfinished guessword. Returns number of characters matched.
Also, returns zero if the character is already guessed. */
int letterFill(char guess, string secretword, string &guessword)
{
	int i;
	int matches = 0;
	int len = secretword.length();
	for (i = 0; i< len; i++)
	{
		// Did we already match this letter in a previous guess?
		if (guess == guessword[i])
			return 0;
		// Is the guess in the secret word?
		if (guess == secretword[i])
		{
			guessword[i] = guess;
			matches++;
		}
	}
	return matches;
}

